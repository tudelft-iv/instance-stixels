#include "hip/hip_runtime.h"
// This file is part of Instance Stixels:
// https://github.com/tudelft-iv/instance-stixels
//
// Originally, it was part of stixels:
// https://github.com/dhernandez0/stixels
//
// Copyright (c) 2016 Daniel Hernandez Juarez.
// Copyright (c) 2019 Thomas Hehn.
//
// Instance Stixels is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Instance Stixels is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Instance Stixels. If not, see <http://www.gnu.org/licenses/>.

#include "RoadEstimation.h"

RoadEstimation::RoadEstimation()
{}


RoadEstimation::~RoadEstimation()
{}

void RoadEstimation::Initialize(
        const float camera_center_y, const float baseline, const float focal,
        const int rows, const int cols, const int max_dis,
        const float road_vdisparity_threshold) {
    // Get camera parameters
    m_cy = camera_center_y;
    m_b = baseline;
    m_focal = focal;

    // TODO: tune parameters to get better road estimation or use provided
    // vehicle information
    // Default configuration
    m_HoughAccumThr = 25;
    //m_binThr = 0.0f;
    m_binThr = road_vdisparity_threshold;
    m_maxPitch = 50;
    m_minPitch = -50;
    /*
    m_maxCameraHeight = -1.30f;
    m_minCameraHeight = -1.90f;
     */
    m_maxCameraHeight = 1.90f;
    m_minCameraHeight = 1.30f;

    m_maxPitch = m_maxPitch * (float) CV_PI / 180.0f;
    m_minPitch = m_minPitch * (float) CV_PI / 180.0f;
    m_max_dis = max_dis;
    m_rows = rows;
    m_cols = cols;

    m_rho = 0;
    m_theta = 0;
    m_horizonPoint = 0;
    m_pitch = 0;
    m_cameraHeight = 0;

    m_vDisp = (uint8_t*) malloc(m_max_dis * m_rows * sizeof(uint8_t));

    CUDA_CHECK_RETURN(
            hipMalloc((void**) &d_disparity, m_cols*m_rows*sizeof(float)));
    CUDA_CHECK_RETURN(
            hipMalloc((void**) &d_vDisp, m_max_dis * m_rows * sizeof(int)));
    CUDA_CHECK_RETURN(
            hipMalloc((void**) &d_maximum, sizeof(int)));
    CUDA_CHECK_RETURN(
            hipMalloc((void**) &d_vDispBinary,
                       m_max_dis * m_rows * sizeof(uint8_t)));

    m_is_initialized = true;
}

void RoadEstimation::Finish() {
    CUDA_CHECK_RETURN(hipFree(d_vDisp));
    CUDA_CHECK_RETURN(hipFree(d_disparity));
    CUDA_CHECK_RETURN(hipFree(d_maximum));
    CUDA_CHECK_RETURN(hipFree(d_vDispBinary));
    free(m_vDisp);

    m_is_initialized = false;
}

bool RoadEstimation::Compute(const std::vector<pixel_t>& im) {
    CUDA_CHECK_RETURN(
            hipMemcpy(d_disparity,
                       im.data(),
                       im.size() * sizeof(pixel_t),
                       hipMemcpyHostToDevice));

    return Compute(d_disparity);
}

bool RoadEstimation::Compute(pixel_t* d_im){
    pixel_t* d_disparity_tmp = d_im;
    bool ok = false;

    CUDA_CHECK_RETURN(
            hipMemset(d_maximum, 0, sizeof(int)));
    CUDA_CHECK_RETURN(
            hipMemset(d_vDisp, 0, m_max_dis * m_rows * sizeof(int)));

    // Compute the vDisparity histogram
    ComputeHistogram<<<(m_rows*m_cols + 256 - 1)/256, 256>>>(
            d_disparity_tmp, d_vDisp, m_rows, m_cols, m_max_dis);
    ComputeMaximum<<<(m_rows*m_max_dis + 256-1)/256, 256>>>(
            d_vDisp, d_maximum, m_rows, m_max_dis);
    ComputeBinaryImage<<<(m_rows*m_max_dis + 256 - 1)/256, 256>>>(
            d_vDisp, d_vDispBinary, d_maximum, m_binThr, m_rows, m_max_dis);

    // Compute the Hough transform
    float rho, theta, horizonPoint, pitch, cameraHeight, slope;
    if( ComputeHough(d_vDispBinary, rho, theta, horizonPoint, pitch,
                     cameraHeight, slope) ){
        m_rho = rho;
        m_theta = theta;
        m_horizonPoint = (int) ceil(horizonPoint);
        m_pitch = pitch;
        m_cameraHeight = cameraHeight;
        m_slope = slope;
        ok = true;
    }

    return ok;
}

bool RoadEstimation::ComputeHough(
        uint8_t* d_vDispBinary, float& rho, float& theta, float& horizonPoint,
        float& pitch, float& cameraHeight, float& slope) {
    // Compute the Hough transform
    std::vector<cv::Vec2f> lines;
    // Note: When running with rosrun, the next line occasionally throws an
    // illegal memory access error.
    CUDA_CHECK_RETURN(
        hipMemcpy(m_vDisp,
                   d_vDispBinary,
                   m_max_dis * m_rows * sizeof(uint8_t),
                   hipMemcpyDeviceToHost));
    cv::Mat vDisp(m_rows, m_max_dis, CV_8UC1, m_vDisp);
    cv::HoughLines(vDisp, lines, 1.0, CV_PI/180, m_HoughAccumThr);

    // Get the best line from hough
    for (size_t i=0; i < lines.size(); i++) {
        // Get rho and theta
        rho = abs(lines[i][0]);
        theta = lines[i][1];

        // Compute camera position
        ComputeCameraProperties(vDisp, rho, theta, horizonPoint, pitch,
                                cameraHeight, slope);

        //printf("%f (%f %f) %f (%f %f)\n",
        //       pitch, m_minPitch, m_maxPitch, cameraHeight,
        //       m_minCameraHeight, m_maxCameraHeight);
        //if (pitch >= m_minPitch &&
        //    pitch <= m_maxPitch &&
        //    cameraHeight >= m_minCameraHeight &&
        //    cameraHeight <= m_maxCameraHeight) {
        if (pitch>=m_minPitch && pitch<=m_maxPitch) {
            return true;
        }
    }

    return false;
}

void RoadEstimation::ComputeCameraProperties(
        cv::Mat vDisp, const float rho, const float theta, float& horizonPoint,
        float& pitch, float& cameraHeight, float& slope) const {
    // Compute Horizon Line (2D)
    horizonPoint = rho/sinf(theta);

    // Compute pitch -> arctan((cy - y0Hough)/focal)
    // It is negative because y axis is inverted.
    pitch = -atanf((m_cy - horizonPoint)/(m_focal));

    // Compute the slope needed to compute the Camera height.
    float last_row = (float) (vDisp.rows-1);
    float vDispDown = (rho - last_row*sinf(theta)) / cosf(theta);
    slope = (0 - vDispDown) / (horizonPoint - last_row);

    // Compute the camera height -> baseline*cos(pitch)/slopeHough
    cameraHeight = m_b * cosf(pitch) / slope;
}

