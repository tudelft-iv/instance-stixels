#include "hip/hip_runtime.h"
// This file is part of Instance Stixels:
// https://github.com/tudelft-iv/instance-stixels
//
// Originally, it was part of stixels:
// https://github.com/dhernandez0/stixels
//
// Copyright (c) 2016 Daniel Hernandez Juarez.
// Copyright (c) 2019 Thomas Hehn.
//
// Instance Stixels is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Instance Stixels is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Instance Stixels. If not, see <http://www.gnu.org/licenses/>.

#include "StixelsKernels.h"
#include <cassert>

#define MIN_STIXEL_HEIGHT 1
// TODO: remove unless ever used again (21-10-19)
// 0 when h5 only includes u,v; 1 when median disparity is also in h5
#define UV_OFFSET 0

__inline__ __device__ float FastLog(const float v) {
    return __logf(v);
}

__inline__ __device__ float NegFastLogDiv(const float v, const float v2) {
    //return -__logf(v/v2);
    return -__logf(v) + __logf(v2);
}

__inline__ __device__ float GetPriorCost(const int vB, const int rows) {
    return NegFastLogDiv(1.0f, (float)(rows-vB));
}

// Given array = [1, ..., 8] the result is
// cumsum = [0, 1, 3, 6, ..., 28]
// Note: vB and vT belong to stixel! See also ComputePrefixSum.
__inline__ __device__ pixel_t ComputeMean(
        const int vB, const int vT, const pixel_t *d_sum,
        const pixel_t *d_valid, const float invalid_disparity){
    pixel_t mean = 0;
    if(invalid_disparity >= 0){
        const pixel_t valid_dif = d_valid[vT+1]-d_valid[vB];
        mean = (valid_dif == 0) ? 0 : (d_sum[vT+1]-d_sum[vB])/valid_dif;
    }
    else{
        mean = (d_sum[vT+1]-d_sum[vB])/(vT+1-vB);
    }

    return mean;
}

__inline__ __device__ pixel_t ComputeNonInstanceOffsetCost(
        const int vB, const int vT,
        const int32_t* instance_offsetsx_ps, // __restricted__?
        const int32_t* instance_offsetsy_ps){ // __restricted__?
    // Offset of 0 is ideal.
    float cost = DownsampledSum(instance_offsetsx_ps, vB, vT)
                 + DownsampledSum(instance_offsetsy_ps, vB, vT);
    return cost;
}

__inline__ __device__ pixel_t ComputeInstanceOffsetCost(
        const int vB, const int vT,
        const int64_t* instance_meansx_ps, // TODO: __restricted__?
        const int64_t* instance_meansy_ps,
        const int64_t* instance_meansx2_ps,
        const int64_t* instance_meansy2_ps){
    const float meanx = instance_meansx_ps[vT+1] - instance_meansx_ps[vB];
    const float meany = instance_meansy_ps[vT+1] - instance_meansy_ps[vB];
    const float meanx2 = instance_meansx2_ps[vT+1] - instance_meansx2_ps[vB];
    const float meany2 = instance_meansy2_ps[vT+1] - instance_meansy2_ps[vB];
    const float height = vT+1.0-vB;
    float cost = meanx2 - meanx*meanx/height + meany2 - meany*meany/height;

    return cost;
}

__inline__ __device__ float GetPriorCostSkyFromObject(
        pixel_t previous_mean, const float epsilon, const float prior_cost) {
    float cost = logf(2.0f)+prior_cost;

    if(previous_mean < epsilon) {
        cost = MAX_LOGPROB;
    }
    return cost;
}

__inline__ __device__ float GetPriorCostSkyFromGround(
        const int vB, float *ground_function, const float prior_cost) {
    const int previous_vT = vB-1;

    const float prev_gf = ground_function[previous_vT];
    const float cost = (prev_gf < 1.0f) ? prior_cost : MAX_LOGPROB;

    return cost;
}

__inline__ __device__ float ComputeObjectDisparityRange(
        const float previous_mean, const float baseline,
        const float focal, const float range_objects_z) {
    float range_disp = 0;
    if(previous_mean != 0) {
        const float pmean_plus_z =
            (baseline * focal / previous_mean) + range_objects_z;
        range_disp = previous_mean - (baseline * focal / pmean_plus_z);
    }
    return range_disp;
}

__inline__ __device__ float GetPriorCostObjectFromGround(
        const int vB, float fn, const float max_disf,
        const float *ground_function, const float prior_cost,
        const float epsilon, const float pgrav, const float pblg) {
    float cost = -logf(0.7f) + prior_cost;

    const int previous_vT = vB-1;
    float fn_previous = ground_function[previous_vT];
    if(fn_previous < 0.0f) {
        fn_previous = 0.0f;
    }

    if(fn > (fn_previous+epsilon)) {
        // It should not be 0, fn_previous could be almost m_max_dis-1
        // but m_epsilon should be small
        cost += NegFastLogDiv(pgrav, max_disf-fn_previous-epsilon);
    } else if(fn < (fn_previous-epsilon)) {
        // fn >= 0 then previous_mean-dif_dis > 0
        const float pmean_sub = fn_previous - epsilon;
        cost += NegFastLogDiv(pblg, pmean_sub);
    } else {
        cost += NegFastLogDiv(1.0f - pgrav - pblg, 2.0f * epsilon);
    }
    return cost;
}

__inline__ __device__ float GetPriorCostObjectFromObject(
        const int vB, const float fn, const pixel_t previous_mean,
        const float *object_disparity_range, const int vhor,
        const float max_disf, const float pord, const float prior_cost) {
    const int previous_vT = vB - 1;
    float cost = (previous_vT < vhor) ? -logf(0.7f) : logf(2.0f);
    cost += prior_cost;

    float dif_dis = object_disparity_range[(int) previous_mean];
    if(dif_dis < 0.0f) {
        dif_dis = 0.0f;
    }

    if(fn > (previous_mean + dif_dis)) {
        // It should not be 0, previous_mean could be almost m_max_dis-1
        // but dif_dis should be small
        cost += NegFastLogDiv(pord, max_disf - previous_mean - dif_dis);
    } else if(fn < (previous_mean - dif_dis)) {
        // fn >= 0 then previous_mean-dif_dis > 0
        const float pmean_sub = previous_mean - dif_dis;
        cost += NegFastLogDiv(1.0f - pord, pmean_sub);
    } else {
        cost = MAX_LOGPROB;
    }
    return cost;
}

__inline__ __device__ float GetPriorCostObjectFromSky(
        const float fn, const float max_disf,
        const float prior_cost, const float epsilon) {
    float cost = MAX_LOGPROB;

    if(fn > epsilon) {
        cost = NegFastLogDiv(1.0f, max_disf - epsilon) + prior_cost;
    }

    return cost;
}

__inline__ __device__ float GetPriorCostGround(const float prior_cost) {
    return -logf(0.3f)+prior_cost;
}

__inline__ __device__ float GetPriorCostObjectFirst(
        const bool below_vhor_vT, const float rows_log,
        const float max_dis_log) {
    const float pvt = below_vhor_vT ? logf(2.0f) : 0.0f;
    return rows_log + pvt + max_dis_log;
}

__inline__ __device__ float GetPriorCostGroundFirst(const float rows_log) {
    // Only below horizon
    return logf(2.0f) + rows_log;
}

__inline__ __device__ float GetDataCostSky(
        const pixel_t d, const float pnexists_given_sky_log,
        const float normalization_sky, const float inv_sigma2_sky,
        const float puniform_sky, const float nopnexists_given_sky_log,
        const float invalid_disparity) {

    float data_cost = pnexists_given_sky_log;
    if(d != invalid_disparity) {
        const float pgaussian = normalization_sky + d*d*inv_sigma2_sky;

        const float p_data = fminf(puniform_sky, pgaussian);
        data_cost = p_data+nopnexists_given_sky_log;
    }
    return data_cost;
}

__inline__ __device__ float GetDataCostGround(
        const float fn, const int v,
        const pixel_t d, const float pnexists_given_ground_log,
        const float* normalization_ground, const float* inv_sigma2_ground,
        const float puniform, const float nopnexists_given_ground_log,
        const float invalid_disparity) {

    float data_cost = pnexists_given_ground_log;
    if(d != invalid_disparity) {
        const float model_diff = (d-fn);
        const float pgaussian = normalization_ground[v]
                                + model_diff*model_diff*inv_sigma2_ground[v];

        const float p_data = fminf(puniform, pgaussian);
        data_cost = p_data + nopnexists_given_ground_log;
    }
    return data_cost;
}

__inline__ __device__ float warp_prefix_sum(
        const int i, const int fn, const pixel_t* __restrict__ d_disparity,
        const float* __restrict__ d_obj_cost_lut,
        const StixelParameters params, float *s_data, const float add) {
    const int lane = threadIdx.x % WARP_SIZE;
    const int col = blockIdx.x;

    // Lookup the cost for current disparity value.
    int dis = 0;
    if(i+lane < params.rows) {
        dis = (int) d_disparity[col*params.rows+i+lane];
    }
    float cost = d_obj_cost_lut[fn*params.max_dis+dis];
    if(lane == 0) {
        cost += add;
    }

    // Parallel prefix sum logic.
#pragma unroll
    for (int j = 1; j < WARP_SIZE; j *= 2) {
#if (__CUDA_ARCH__ < 700)
        float n = __shfl_up(cost, j);
#else
        float n = __shfl_up_sync(0xFFFFFFFF, cost, j);
#endif

        if (lane >= j) cost += n;
    }

    // TODO: isn't the +1 terrible in terms of memory access?
    s_data[i+lane+1] = cost;

#if (__CUDA_ARCH__ < 700)
    return __shfl(cost, WARP_SIZE-1);
#else
    return __shfl_sync(0xFFFFFFFF, cost, WARP_SIZE-1);
#endif
}

__inline__ __device__ void ComputePrefixSumWarp2(
        const int fn, const pixel_t* __restrict__ d_disparity,
        const float* __restrict__ d_obj_cost_lut,
        const StixelParameters params, float *arr,
        const int n, const int n_power2) {
    float add = 0.0f;
    const int lane = threadIdx.x % WARP_SIZE;

    if(lane == 0) {
        arr[0] = 0.0f;
    }

    // Apply warp_prefix_sum to different sections of "arr", which is a vector
    // for a single fn value.
    // First section: 0 to (warp_size-1), 2nd: warp_size to (2*warp_size-1),...
    // NOTE: The sections will write to
    // arr[1 to warp_size] and arr[warp_size+1 to 2*warp_size]
    for(int i = 0; i < n_power2; i += WARP_SIZE) {
        add = warp_prefix_sum(i, fn, d_disparity, d_obj_cost_lut,
                              params, arr, add);
    }
}

template <bool PAIRWISE>
__global__ void StixelsKernel(
        const pixel_t* __restrict__ d_disparity,
        int32_t* __restrict__ d_segmentation, // TODO: __restrict__ ? profile!
        const StixelParameters params,
        const float* __restrict__ d_ground_function,
        const float* __restrict__ d_normalization_ground,
        const float* __restrict__ d_inv_sigma2_ground,
        const float* __restrict__ d_object_disparity_range,
        const float* __restrict__ d_object_lut,
        Section* __restrict__ d_stixels,
        float* d_instance_centerofmass,
        int32_t* d_instance_indices,
        bool* d_instance_core_candidates,
        int32_t* d_instances_per_class,
        int64_t* d_instance_meansx_ps){
    const int col = blockIdx.x;
    const int row = threadIdx.x;
    // Note this is required at the moment. Making it work in some other
    // will likely result in performance drawbacks.
    assert(DOWNSAMPLE_FACTOR == params.column_step);

    extern __shared__ int s[];
    // --- Divide shared memory into separate arrays.
    // sky_lut                  = sizeof(float)   * rows_power2
    // ground_lut               = sizeof(float)   * rows_power2
    // ground_function          = sizeof(float)   * rows_power2
    // object_disparity_range   = sizeof(float)   * max_dis
    // cost_table               = sizeof(float)   * rows_power2*3
    // index_table              = sizeof(int16_t) * rows_power2*3
    // disparity_prefixsum      = sizeof(pixel_t) * rows_power2
    // valid_disparity          = sizeof(pixel_t) * rows_power2
    // instance_offsetsx        = sizeof(int64_t) * rows_power2
    // instance_offsetsy        = sizeof(int64_t) * rows_power2
    // instance_meansx_ps       = sizeof(int64_t) * rows_power2
    // instance_meansy_ps       = sizeof(int64_t) * rows_power2
    // NULL;
    float *sky_lut = (float*)&s;
    float *ground_lut = &sky_lut[params.rows_power2];
    float *ground_function = &ground_lut[params.rows_power2];
    float *object_disparity_range = &ground_function[params.rows_power2];
    float *cost_table = &object_disparity_range[params.max_dis];
    int16_t *index_table = (int16_t*) &cost_table[params.rows_power2*3];
    pixel_t *disparity_prefixsum =
        (pixel_t*) &index_table[params.rows_power2*3];
    pixel_t *valid_disparity =
        (pixel_t*) &disparity_prefixsum[params.rows_power2];

    // Shared memory is limited for non-volta cards.
    int64_t* instance_meansx_ps = NULL;
    if(d_instance_meansx_ps == NULL){
        instance_meansx_ps =
            (int64_t*) &valid_disparity[params.rows_power2];
    }
    else{
        instance_meansx_ps =
            &d_instance_meansx_ps[col * params.rows_power2 * 4];
    }
    int64_t* instance_meansy_ps =
        (int64_t*) &instance_meansx_ps[params.rows_power2];
    int64_t* instance_meansx2_ps =
        (int64_t*) &instance_meansy_ps[params.rows_power2];
    int64_t* instance_meansy2_ps =
        (int64_t*) &instance_meansx2_ps[params.rows_power2];
    int32_t* instance_offsetsx_ps = NULL;
    int32_t* instance_offsetsy_ps = NULL;

    const float prior_weight = params.prior_weight;
    const float disparity_weight = params.disparity_weight;
    const float segmentation_weight = params.segmentation_weight;

    assert(params.rows % MIN_STIXEL_HEIGHT == 0);

    if(row < params.rows) {
        const pixel_t d = d_disparity[col*params.rows + row];

        cost_table[row] = MAX_LOGPROB;
        cost_table[params.rows + row] = MAX_LOGPROB;
        cost_table[2*params.rows + row] = MAX_LOGPROB;

        if(row < params.max_dis) {
            object_disparity_range[row] = d_object_disparity_range[row];
        }

        if(params.invalid_disparity >= 0){
            const int va = d != params.invalid_disparity;
            valid_disparity[row] = (pixel_t) va;
            disparity_prefixsum[row] = ((pixel_t) va) * d;
        }
        else{
            disparity_prefixsum[row] = d;
        }

        // --- Load instance data into shared memory.
        // Skip semantic channels indices.
        const int row_index =
            col * params.segmentation_channels
                * params.rows_power2_segmentation
            + (params.segmentation_classes + UV_OFFSET)
                * params.rows_power2_segmentation
            + row/DOWNSAMPLE_FACTOR;

        // Note: input considers origin at top left and y pointing DOWN.
        instance_meansx_ps[row] =
                ((params.column_step * col + 0.5 * (params.column_step-1.0))
                + d_segmentation[row_index + params.rows_power2_segmentation] + 0.5);
        instance_meansy_ps[row] =
                (row - d_segmentation[row_index] + 0.5);
        instance_meansx2_ps[row] =
            (instance_meansx_ps[row] * instance_meansx_ps[row]);
        instance_meansy2_ps[row] =
            (instance_meansy_ps[row] * instance_meansy_ps[row]);

        if(row % DOWNSAMPLE_FACTOR == 0){
            d_segmentation[row_index + params.rows_power2_segmentation] *=
                    d_segmentation[row_index + params.rows_power2_segmentation];
            d_segmentation[row_index] *=
                    d_segmentation[row_index];
        }
        // TODO: actually squared offsets! refactor
        instance_offsetsx_ps = &d_segmentation[row_index - row/DOWNSAMPLE_FACTOR
                                               + params.rows_power2_segmentation];
        instance_offsetsy_ps = &d_segmentation[row_index - row/DOWNSAMPLE_FACTOR];

        // sky_lut[row < params.vhor] will not be used, except when computing
        // prefix sum.
        sky_lut[row] =
            (row < params.vhor) ?
            0 : //MAX_LOGPROB :
            GetDataCostSky(d,
                           params.pnexists_given_sky_log,
                           params.normalization_sky,
                           params.inv_sigma2_sky,
                           params.puniform_sky,
                           params.nopnexists_given_sky_log,
                           params.invalid_disparity);

        ground_function[row] = d_ground_function[row];
        const float gf = ground_function[row];
        ground_lut[row] =
            (row >= params.vhor) ?
            MAX_LOGPROB :
            GetDataCostGround(gf, row, d,
                              params.pnexists_given_ground_log,
                              d_normalization_ground,
                              d_inv_sigma2_ground,
                              params.puniform,
                              params.nopnexists_given_ground_log,
                              params.invalid_disparity);

        // Reason: Usage of "column" in the precomputation of Object LUT and
        //          need writes to luts before ComputePrefixSum
        __syncthreads();

        if(params.invalid_disparity >= 0){
            ComputePrefixSum(valid_disparity, params.rows_power2);
        }
        ComputePrefixSum(disparity_prefixsum, params.rows_power2);
        ComputePrefixSum(instance_meansx_ps, params.rows_power2);
        ComputePrefixSum(instance_meansy_ps, params.rows_power2);
        ComputePrefixSum(instance_meansx2_ps, params.rows_power2);
        ComputePrefixSum(instance_meansy2_ps, params.rows_power2);
        ComputePrefixSum(ground_lut, params.rows_power2);
        ComputePrefixSum(sky_lut, params.rows_power2);
        for(int c = 0; c < params.segmentation_classes + 2; c++){
            // Note: We can use ComputePrefixSum after JoinColumnsSegmentation.
            ComputePrefixSum(&d_segmentation[
                                col * params.rows_power2_segmentation
                                    * params.segmentation_channels
                                + c * params.rows_power2_segmentation],
                             params.rows_power2_segmentation);
        }

        // only if constexpr PAIRWISE, thus "unused"
        __attribute__((unused)) const float max_disf = (float) params.max_dis;

        const int vT = row;
        const int obj_data_idx = col * (params.rows_power2+1) * params.max_dis;

        // First segment: Special case vB = 0
        // TODO: Try to remove this by assuming that we have a ground/road
        // stixel first.
        __syncthreads();
        if(vT % MIN_STIXEL_HEIGHT == MIN_STIXEL_HEIGHT-1){
        {
            const int vB = 0;
            // only !PAIRWISE
            __attribute__((unused)) const float inverse_height = 1./(vT+1-vB);

            // --- Compute instance term.
            // Compute difference from instance stixel means.
            const float instance_cost = params.instance_weight *
                ComputeInstanceOffsetCost(vB, vT,
                                          instance_meansx_ps,
                                          instance_meansy_ps,
                                          instance_meansx2_ps,
                                          instance_meansy2_ps);
            // Compute difference from pixel positions.
            const float non_instance_cost = params.instance_weight *
                ComputeNonInstanceOffsetCost(vB, vT,
                                             instance_offsetsx_ps,
                                             instance_offsetsy_ps);

            // Min ground semantic class
            const float cost_ground_segmentation = GetGroundSegmentationCost(
                    &d_segmentation[col * params.rows_power2_segmentation
                                    * params.segmentation_channels],
                    vB, vT, params.rows_power2_segmentation)
                    + non_instance_cost;
            // Min object semantic class
            const float cost_object_segmentation = GetObjectSegmentationCost(
                    &d_segmentation[col * params.rows_power2_segmentation
                                    * params.segmentation_channels],
                    vB, vT, params.rows_power2_segmentation,
                    instance_cost, non_instance_cost);

            // Compute disparity data terms.
            pixel_t obj_fn =
                ComputeMean(vB, vT, disparity_prefixsum, valid_disparity,
                            params.invalid_disparity);
            // Sometimes obj_fni is negative (~ -1e-5). This means that in the
            // prefix sum, there is a value which is larger that its
            // predecessor. As all entries in "sum" are positive (disparities),
            // this should not happen (I checked this.). I think this an
            // numeric issue of the "ComputePrefixSum" sum.
            // NOTE: This also means that ground_lut and sky_lut might suffer
            // from the same problem.
            if(obj_fn < 0) {
                obj_fn = 0;
            }
            const int obj_fni = (int) floorf(obj_fn);

            const float cost_ground_data =
                ground_lut[vT+1] - ground_lut[vB];
            // NOTE: d_object_lut depends on disparity image, whereas
            // d_obj_cost_lut is precomputed and does not depend on current
            // disparity image.
            const float cost_object_data =
                d_object_lut[obj_data_idx+obj_fni*(params.rows_power2+1) +vT+1]
                -d_object_lut[obj_data_idx+obj_fni*(params.rows_power2+1) +vB];

            // Compute priors costs
            const int index_pground = vT*3 + GROUND;
            const int index_pobject = vT*3 + OBJECT;
            const bool below_vhor_vT = vT <= params.vhor;

            // Ground
            if(below_vhor_vT) {
                const float curr_cost_ground = cost_table[index_pground];
                float cost_ground;
                if constexpr(PAIRWISE){
                    const float cost_ground_prior =
                        GetPriorCostGroundFirst(params.rows_log);
                    cost_ground =
                        disparity_weight * cost_ground_data
                        + prior_weight * cost_ground_prior
                        + segmentation_weight * cost_ground_segmentation;
                }
                else{
                    cost_ground =
                        disparity_weight * cost_ground_data
                        + prior_weight * inverse_height
                        + segmentation_weight * cost_ground_segmentation;
                }
                if( cost_ground < curr_cost_ground ) {
                    cost_table[index_pground] = cost_ground;
                    index_table[index_pground] = GROUND;
                }
            }

            // Object
            const float curr_cost_object = cost_table[index_pobject];
            float cost_object;
            if constexpr(PAIRWISE){
                const float cost_object_prior =
                    GetPriorCostObjectFirst(below_vhor_vT, params.rows_log,
                                            params.max_dis_log);
                cost_object =
                    disparity_weight * cost_object_data
                    + prior_weight * cost_object_prior
                    + segmentation_weight * cost_object_segmentation;
            }
            else{
                cost_object =
                    disparity_weight * cost_object_data
                    + prior_weight * inverse_height
                    + segmentation_weight * cost_object_segmentation;
            }
            if( cost_object < curr_cost_object ) {
                cost_table[index_pobject] = cost_object;
            }
            // Since OBJECT is the fallback class, this should not remain
            // uninitialized.
            // index_table[...] < 3 will cause backtracing to finish.
            index_table[index_pobject] = OBJECT;
        }
        } // if(vT % MIN_STIXEL_HEIGHT == 0)

        // Computing cases vB > 0
        // e.g. MIN_STIXEL_HEIGHT = 8:
        // vB = 0, 8, 16, 24, 32, ...
        // vT = 7, 15, 23, 31, 43, ...
        for(int vB = MIN_STIXEL_HEIGHT;
                vB < params.rows;
                vB += MIN_STIXEL_HEIGHT) {
            __syncthreads();

            // Skip cases where vT (= row = threadIdx.x) is larger than vB
            // Note: insert minimum stixel size here as vB + min_size.
            if(vT >= vB && vT % MIN_STIXEL_HEIGHT == MIN_STIXEL_HEIGHT-1) {
                __attribute__((unused)) const float inverse_height = 1./(vT+1-vB);
                // --- Compute instance term.
                // Compute difference from instance stixel means.
                const float instance_cost = params.instance_weight *
                    ComputeInstanceOffsetCost(vB, vT,
                                              instance_meansx_ps,
                                              instance_meansy_ps,
                                              instance_meansx2_ps,
                                              instance_meansy2_ps);
                // Compute difference from pixel positions.
                const float non_instance_cost = params.instance_weight *
                    ComputeNonInstanceOffsetCost(vB, vT,
                                                 instance_offsetsx_ps,
                                                 instance_offsetsy_ps);

                // --- Compute semantics.
                // Min ground semantic class
                const float cost_ground_segmentation =
                    GetGroundSegmentationCost(
                        &d_segmentation[col * params.rows_power2_segmentation
                                        * params.segmentation_channels],
                        vB, vT, params.rows_power2_segmentation)
                    + non_instance_cost;
                // Min object semantic class
                const float cost_object_segmentation =
                    GetObjectSegmentationCost(
                        &d_segmentation[col * params.rows_power2_segmentation
                                        * params.segmentation_channels],
                        vB, vT, params.rows_power2_segmentation,
                        instance_cost, non_instance_cost);
                // Min object semantic class
                const float cost_sky_segmentation =
                    GetSkySegmentationCost(
                        &d_segmentation[col * params.rows_power2_segmentation
                                        * params.segmentation_channels],
                        vB, vT, params.rows_power2_segmentation)
                    + non_instance_cost;

                // Compute disparity data term
                pixel_t obj_fn =
                    ComputeMean(vB, vT, disparity_prefixsum, valid_disparity,
                                params.invalid_disparity);
                // See obj_fni above.
                if(obj_fn < 0) {
                    obj_fn = 0;
                }
                const int obj_fni = (int) floorf(obj_fn);

                const float cost_object_data =
                    d_object_lut[obj_data_idx
                                 + obj_fni * (params.rows_power2+1)
                                 + vT+1]
                    - d_object_lut[obj_data_idx
                                   + obj_fni * (params.rows_power2+1)
                                   + vB];
                // Uniform distribution over remaining rows.
                __attribute__((unused)) float prior_cost = 0;
                if constexpr(PAIRWISE){
                    prior_cost = GetPriorCost(vB, params.rows);
                }

                // Cost for previous_vT has already been computed since
                // vT >= vB and syncthreads call above.
                const int previous_vT = vB-1;
                const bool below_vhor_vTprev = previous_vT < params.vhor;

                __attribute__((unused)) pixel_t previous_mean = 0;
                if constexpr(PAIRWISE){
                    const int previous_object_vB =
                        index_table[previous_vT*3 + OBJECT] / 3;
                    previous_mean =
                        ComputeMean(previous_object_vB, previous_vT,
                                    disparity_prefixsum, valid_disparity,
                                    params.invalid_disparity);
                    if(previous_mean < 0) {
                        previous_mean = 0;
                    }
                }

                if(below_vhor_vTprev) { // previous_vT < parames.vhor
                    // Ground
                    const float cost_ground_data =
                        ground_lut[vT+1] - ground_lut[vB];
                    const int index_pground = vT*3 + GROUND;

                    const float curr_cost_ground = cost_table[index_pground];
                    float cost_ground_prior1 =
                        cost_table[previous_vT*3 + GROUND];
                    float cost_ground_prior2 =
                        cost_table[previous_vT*3 + OBJECT];

                    float cost_ground;
                    if constexpr(PAIRWISE){
                        const float prev_cost = GetPriorCostGround(prior_cost);
                        cost_ground_prior1 +=
                            prior_weight * prev_cost;
                        cost_ground_prior2 +=
                            prior_weight * prev_cost;
                        const float cost_ground_minprior =
                            fminf(cost_ground_prior1, cost_ground_prior2);
                        cost_ground =
                            disparity_weight * cost_ground_data
                            + prior_weight * cost_ground_minprior
                            + segmentation_weight * cost_ground_segmentation;
                    }
                    else{
                        const float cost_ground_minprior =
                            fminf(cost_ground_prior1, cost_ground_prior2);
                        cost_ground =
                            disparity_weight * cost_ground_data
                            + prior_weight * inverse_height
                            + segmentation_weight * cost_ground_segmentation;
                    }
                    if( cost_ground < curr_cost_ground ) {
                        cost_table[index_pground] = cost_ground;
                        int min_prev = OBJECT;
                        if(cost_ground_prior1 < cost_ground_prior2) {
                            min_prev = GROUND;
                        }
                        index_table[index_pground] = vB*3 + min_prev;
                    }
                } else { // previous_vT (=vB-1) >= params.vhor
                    // Sky
                    const float cost_sky_data = sky_lut[vT+1] - sky_lut[vB];
                    const int index_psky = vT*3 + SKY;

                    const float curr_cost_sky = cost_table[index_psky];
                    float cost_sky_prior1 =
                        cost_table[previous_vT*3 + GROUND];
                    float cost_sky_prior2 =
                        cost_table[previous_vT*3 + OBJECT];
                    float cost_sky;
                    if constexpr(PAIRWISE){
                        cost_sky_prior1 +=
                            prior_weight *
                            GetPriorCostSkyFromGround(vB, ground_function,
                                                      prior_cost);

                        cost_sky_prior2 +=
                            prior_weight *
                            GetPriorCostSkyFromObject(previous_mean,
                                                      params.epsilon, prior_cost);
                        const float cost_sky_minprior =
                            fminf(cost_sky_prior1, cost_sky_prior2);

                        cost_sky =
                            disparity_weight * cost_sky_data
                            + prior_weight * cost_sky_minprior
                            + segmentation_weight * cost_sky_segmentation;
                    }
                    else{
                        const float cost_sky_minprior =
                            fminf(cost_sky_prior1, cost_sky_prior2);

                        cost_sky =
                            disparity_weight * cost_sky_data
                            + prior_weight * inverse_height
                            + segmentation_weight * cost_sky_segmentation;
                    }
                    if( cost_sky < curr_cost_sky ) {
                        cost_table[index_psky] = cost_sky;
                        int min_prev = OBJECT;
                        if(cost_sky_prior1 < cost_sky_prior2) {
                            min_prev = GROUND;
                        }
                        index_table[index_psky] = vB*3 + min_prev;
                    }
                }

                // Object
                const int index_pobject = vT*3+OBJECT;

                const float curr_cost_object = cost_table[index_pobject];
                float cost_object;
                float cost_object_prior1 =
                    cost_table[previous_vT*3+GROUND];
                float cost_object_prior2 =
                    cost_table[previous_vT*3 + OBJECT];
                float cost_object_prior3 =
                    cost_table[previous_vT*3 + SKY];
                if constexpr(PAIRWISE){
                    cost_object_prior1 +=
                        prior_weight *
                        GetPriorCostObjectFromGround(vB, obj_fn, max_disf,
                                                     ground_function, prior_cost,
                                                     params.epsilon,
                                                     params.pgrav, params.pblg);

                    cost_object_prior2 +=
                        prior_weight *
                        GetPriorCostObjectFromObject(vB, obj_fn, previous_mean,
                                                     object_disparity_range,
                                                     params.vhor, max_disf,
                                                     params.pord, prior_cost);
                    cost_object_prior3 +=
                        prior_weight *
                        GetPriorCostObjectFromSky(obj_fn, max_disf, prior_cost,
                                                  params.epsilon);
                    const float cost_object_minprior =
                        fminf( fminf(cost_object_prior1, cost_object_prior2),
                               cost_object_prior3);

                    cost_object =
                        disparity_weight * cost_object_data
                        + prior_weight * cost_object_minprior
                        + segmentation_weight * cost_object_segmentation;
                }
                else{
                    const float cost_object_minprior =
                        fminf( fminf(cost_object_prior1, cost_object_prior2),
                               cost_object_prior3);

                    cost_object =
                        disparity_weight * cost_object_data
                        + prior_weight * inverse_height
                        + segmentation_weight * cost_object_segmentation;
                }

                if( cost_object < curr_cost_object ) {
                    cost_table[index_pobject] = cost_object;
                    int min_prev = OBJECT;
                    if(cost_object_prior1 < cost_object_prior2) {
                        min_prev = GROUND;
                    }
                    if(cost_object_prior3 <
                            fminf(cost_object_prior1, cost_object_prior2)) {
                        min_prev = SKY;
                    }
                    index_table[index_pobject] = vB*3 + min_prev;
                }
            } // if(vT >= vB && vT % MIN_STIXEL_HEIGHT == 0) {
        }

        __syncthreads();

        // Backtracing
        if(row == 0) {
            int vT = params.rows-1;
            const float last_ground = cost_table[vT*3 + GROUND];
            const float last_object = cost_table[vT*3 + OBJECT];
            const float last_sky = cost_table[vT*3 + SKY];

            // OBJECT is the fallback geometric class. That means that in case all
            // geometric class have the same cost (most probably == MAX_LOGPROB),
            // the stixel will cover the entire column and have geometric class
            // OBJECT.
            int type = OBJECT;

            if(last_ground < last_object) {
                type = GROUND;
            }
            if(last_sky < fminf(last_ground, last_object)) {
                type = SKY;
            }
            int min_idx = vT*3 + type;

            int prev_vT;
            int i = 0;
            do {
                prev_vT = (index_table[min_idx] / 3) - 1;
                Section sec;
                sec.vT = vT;
                sec.type = type;
                sec.vB = prev_vT + 1;
                sec.disparity =
                    (float) ComputeMean(sec.vB, sec.vT, disparity_prefixsum,
                                        valid_disparity,
                                        params.invalid_disparity);
                sec.cost = fminf(cost_table[sec.vT*3+type], 1e4);
                sec.instance_meanx = float(instance_meansx_ps[sec.vT+1]
                                       - instance_meansx_ps[sec.vB])
                                     / (sec.vT+1-sec.vB);
                sec.instance_meany = float(instance_meansy_ps[sec.vT+1]
                                       - instance_meansy_ps[sec.vB])
                                     / (sec.vT+1-sec.vB);

                if(sec.type == GROUND){
                    sec.semantic_class =
                        GetGroundSegmentationClass(
                            &d_segmentation[col
                                            * params.rows_power2_segmentation
                                            * params.segmentation_channels],
                            sec.vB, sec.vT, params.rows_power2_segmentation);
                }
                // TODO: should we then also change sec.type
                // from OBJECT to SKY for disparity < 1.0??
                else if(sec.type == SKY || sec.disparity < 1.0){
                    sec.type = SKY;
                    sec.semantic_class =
                        GetSkySegmentationClass(
                            &d_segmentation[col
                                            * params.rows_power2_segmentation
                                            * params.segmentation_channels],
                            sec.vB, sec.vT, params.rows_power2_segmentation);
                }
                else{ // OBJECT
                    // --- Compute instance term.
                    // Compute difference from instance stixel means.
                    const float instance_cost = params.instance_weight *
                        ComputeInstanceOffsetCost(sec.vB, sec.vT,
                                                  instance_meansx_ps,
                                                  instance_meansy_ps,
                                                  instance_meansx2_ps,
                                                  instance_meansy2_ps);
                    // Compute difference from pixel positions.
                    const float non_instance_cost = params.instance_weight *
                        ComputeNonInstanceOffsetCost(sec.vB, sec.vT,
                                                     instance_offsetsx_ps,
                                                     instance_offsetsy_ps);

                    sec.semantic_class =
                        GetObjectSegmentationClass(
                            &d_segmentation[col
                                            * params.rows_power2_segmentation
                                            * params.segmentation_channels],
                            sec.vB, sec.vT, params.rows_power2_segmentation,
                            instance_cost, non_instance_cost);
                    // TODO: Remove dataset specific constants
                    if(sec.semantic_class >= 11){
                        int instance_idx =
                            atomicAdd(&d_instances_per_class[
                                              sec.semantic_class-11],
                                      1);
                        const int class_offset = (sec.semantic_class-11)
                                                 * params.cols // =realcols
                                                 * params.max_sections;
                        d_instance_centerofmass[(class_offset+instance_idx)*2] =
                            sec.instance_meanx;
                        d_instance_centerofmass[(class_offset+instance_idx)*2+1] =
                            sec.instance_meany;
                        d_instance_indices[(class_offset+instance_idx)*2] = col;
                        d_instance_indices[(class_offset+instance_idx)*2+1] = i;
                        d_instance_core_candidates[class_offset+instance_idx] =
                            (sec.vT+1-sec.vB) >= params.clustering_size_filter;
                    }
                }
                d_stixels[col*params.max_sections + i] = sec;

                type = index_table[min_idx] % 3;
                vT = prev_vT;
                min_idx = prev_vT*3 + type;
                i++;
                assert(i < params.max_sections);
            } while(prev_vT != -1);
            Section sec;
            sec.type = -1;
            d_stixels[col*params.max_sections+i] = sec;
        }
    }
}

__global__ void ComputeObjectLUT(
        const pixel_t* __restrict__ d_disparity,
        const float* __restrict__ d_obj_cost_lut,
        float* __restrict__ d_object_lut,
        const StixelParameters params,
        const int n_power2) {
    const int col = blockIdx.x;
    const int warp_id = threadIdx.x / WARP_SIZE;

    const int blck_step = blockDim.x / WARP_SIZE;
    // Compute prefix sum of costs for different mean disparity values fn.
    // Each warp computes the prefix sum for a couple of fn values (for loop).
    for(int fn = warp_id; fn < params.max_dis; fn += blck_step) {
        ComputePrefixSumWarp2(
                fn, d_disparity, d_obj_cost_lut, params,
                &d_object_lut[col * (params.rows_power2+1) * params.max_dis
                              + fn * (params.rows_power2+1)],
                params.rows, n_power2);
    }
}

__global__ void JoinColumns(
        pixel_t* __restrict__ d_disparity, pixel_t* __restrict__ d_out,
        const int step_size, const bool median,
        const int width_margin, const int rows,
        const int cols, const int real_cols,
        const float invalid_disparity) {
    const int idx = blockIdx.x*blockDim.x+threadIdx.x;
    const int row = idx / real_cols;
    const int col = idx % real_cols;

    if(idx < real_cols*rows) {
        if(median) {
            if(invalid_disparity >= 0){
                pixel_t tmp_row[16];
                int valid_pixels = 0;
                for(int i = 0; i < step_size; i++) {
                    const pixel_t tmp =
                        d_disparity[row*cols + col*step_size + i + width_margin];
                    if(tmp != invalid_disparity){
                        tmp_row[valid_pixels] = tmp;
                        valid_pixels++;
                    }
                }
                // Sort
                // step_size: 8 -> i < 5
                // step_size: 9 -> i < 5
                if(valid_pixels > 0){
                    for(int i = 0; i < (valid_pixels/2)+1; i++) {
                        int min_idx = i;
                        for(int j = i+1; j < valid_pixels; j++) {
                            if(tmp_row[j] < tmp_row[min_idx]) {
                                min_idx = j;
                            }
                        }
                        const pixel_t tmp = tmp_row[i];
                        tmp_row[i] = tmp_row[min_idx];
                        tmp_row[min_idx] = tmp;
                    }
                    pixel_t median = tmp_row[valid_pixels/2];
                    if(valid_pixels % 2 == 0) {
                        // step_size: 8 -> index = 3
                        median = (median+tmp_row[(valid_pixels/2)-1])/2.0f;
                    }
                    d_out[col*rows+rows-row-1] = median;
                }
                else{ // no valid pixels
                    d_out[col*rows+rows-row-1] = invalid_disparity;
                }
            }
            else{ // Invalid disparity < 0
                pixel_t tmp_row[16];
                for(int i = 0; i < step_size; i++) {
                    tmp_row[i] =
                        d_disparity[row*cols + col*step_size + i + width_margin];
                }
                // Sort
                // step_size: 8 -> i < 5
                // step_size: 9 -> i < 5
                for(int i = 0; i < (step_size/2)+1; i++) {
                    int min_idx = i;
                    for(int j = i+1; j < step_size; j++) {
                        if(tmp_row[j] < tmp_row[min_idx]) {
                            min_idx = j;
                        }
                    }
                    const pixel_t tmp = tmp_row[i];
                    tmp_row[i] = tmp_row[min_idx];
                    tmp_row[min_idx] = tmp;
                }
                pixel_t median = tmp_row[step_size/2];
                if(step_size % 2 == 0) {
                    // step_size: 8 -> index = 3
                    median = (median+tmp_row[(step_size/2)-1])/2.0f;
                }
                d_out[col*rows+rows-row-1] = median;
            }
        }
        else { // mean
            pixel_t mean = 0.0f;
// NOTE:
// 1. Computing the mean twice (specifically diving by no. stixels), here and in
// ComputeMean is not correct.
// 2. At the moment, we only consider invalid disparities in case
// the entire row in a column is invalid.
// TODO:
// It should be possible to fix this by counting the number of invalid pixels
// and keeping track of the number of pixels used per stixel. However, this
// will require more memory and processing.
            if(invalid_disparity >= 0){
                int invalid = 0;
                for(int i = 0; i < step_size; i++) {
                    const pixel_t d =
                        d_disparity[row*cols + col*step_size + i + width_margin];
                    if(d != invalid_disparity){
                        mean += d;
                    }
                    else{
                        invalid++;
                    }
                }
                if(invalid != step_size){
                    d_out[col*rows + rows - row-1] = mean / (step_size - invalid);
                }
                else{
                    d_out[col*rows + rows - row-1] = invalid_disparity;
                }
            }
            else{ // Invalid disparity < 0
                for(int i = 0; i < step_size; i++) {
                    mean += d_disparity[row*cols + col*step_size + i + width_margin];
                }
                d_out[col*rows + rows - row-1] = mean / step_size;
            }
        } // closing else from mean/median
    }
}

// Explicit instantiation for binary case!
template __global__ void StixelsKernel<true>(
        const pixel_t* __restrict__ d_disparity,
        int32_t* __restrict__ d_segmentation,
        const StixelParameters params,
        const float* __restrict__ d_ground_function,
        const float* __restrict__ d_normalization_ground,
        const float* __restrict__ d_inv_sigma2_ground,
        const float* __restrict__ d_object_disparity_range,
        const float* __restrict__ d_object_lut,
        Section* __restrict__ d_stixels,
        float* d_instance_centerofmass,
        int32_t* d_instance_indices,
        bool* d_instance_core_candidates,
        int32_t* d_instances_per_class,
        int64_t* d_instance_meansx_ps);
template __global__ void StixelsKernel<false>(
        const pixel_t* __restrict__ d_disparity,
        int32_t* __restrict__ d_segmentation,
        const StixelParameters params,
        const float* __restrict__ d_ground_function,
        const float* __restrict__ d_normalization_ground,
        const float* __restrict__ d_inv_sigma2_ground,
        const float* __restrict__ d_object_disparity_range,
        const float* __restrict__ d_object_lut,
        Section* __restrict__ d_stixels,
        float* d_instance_centerofmass,
        int32_t* d_instance_indices,
        bool* d_instance_core_candidates,
        int32_t* d_instances_per_class,
        int64_t* d_instance_meansx_ps);

